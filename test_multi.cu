
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 5 // Taille des matrices

// Kernel pour la multiplication de matrices
__global__ void matrixMultiplication(int *a, int *b, int *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < n && col < n) {
        int sum = 0;
        for (int i = 0; i < n; ++i) {
            sum += a[row * n + i] * b[i + col * n];
        }
        c[row * n + col] = sum;
    }
}

int main() {
    int *a, *b, *c; // Host matrices
    int *d_a, *d_b, *d_c; // Device matrices

    // Allocation mémoire pour les matrices sur le device
    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**)&d_a, N * N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Erreur lors de l'allocation de mémoire pour d_a: %s\n", hipGetErrorString(cudaStatus));
        return -1;
    }
    cudaStatus = hipMalloc((void**)&d_b, N * N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Erreur lors de l'allocation de mémoire pour d_b: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_a);
        return -1;
    }
    cudaStatus = hipMalloc((void**)&d_c, N * N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Erreur lors de l'allocation de mémoire pour d_c: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_a);
        hipFree(d_b);
        return -1;
    }

    // Allocation mémoire pour les matrices sur l'hôte
    a = (int*)malloc(N * N * sizeof(int));
    b = (int*)malloc(N * N * sizeof(int));
    c = (int*)malloc(N * N * sizeof(int));

    // Initialisation des matrices a et b avec des valeurs aléatoires
    srand(time(NULL));
    for (int i = 0; i < N * N; ++i) {
        a[i] = rand() % 10; // Valeurs entre 0 et 9
        b[i] = rand() % 10; // Valeurs entre 0 et 9
    }

    // Copie des données des matrices de l'hôte au device
    cudaStatus = hipMemcpy(d_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Erreur lors de la copie des données de a au device: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        return -1;
    }
    cudaStatus = hipMemcpy(d_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Erreur lors de la copie des données de b au device: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        return -1;
    }

    // Configuration des dimensions de la grille et du bloc
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Appel du kernel pour la multiplication de matrices
    matrixMultiplication<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // Copie du résultat de la multiplication du device à l'hôte
    cudaStatus = hipMemcpy(c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Erreur lors de la copie du résultat de c au host: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        return -1;
    }

    // Affichage du résultat
    printf("Matrix A:\n");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%d ", a[i * N + j]);
        }
        printf("\n");
    }

    printf("\nMatrix B:\n");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%d ", b[i * N + j]);
        }
        printf("\n");
    }

    printf("\nResult Matrix C:\n");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%d ", c[i * N + j]);
        }
        printf("\n");
    }

    // Libération de la mémoire
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
