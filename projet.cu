#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <malloc.h>
#include <stdlib.h>

#include "mlp.h"

// Kernel pour la multiplication de matrices
__global__ void matrixMultiplicationKernel(double *input_matrix, double *d_weight, double *output_matrix, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < n && col < n) {
        int sum = 0;
        for (int i = 0; i < n; ++i) {
            sum += a[row * n + i] * b[i + col * n];
        }
        c[row * n + col] = sum;
    }
}

typedef struct _neuron NEURON;
struct _neuron {
  int layer;

  double * weight;      // table of weights for incoming synapses
  int nbsynapsesin;     // number of incoming synapses

  NEURON ** synapsesin; // table of pointer to the neurons from
                        // which are coming the synapses
  double bias;

  double value;
  double value_prev;
  double error;
  double error_prev;
};

typedef struct _rnn RNN;
struct _rnn {
  int * layersize;

  int nbneurons;
  NEURON * n;
};

typedef struct _config CONFIG;
struct _config {
  int nbneurons;
  int * layersize;
  int nbsynapses;
  int * synapses;
};


CONFIG * createconfig(int * layersize) {
  CONFIG * conf = (CONFIG*)malloc(sizeof(CONFIG));
  int i;
  conf->nbneurons = 0;
  for(i=1; i<layersize[0]+1; i++) conf->nbneurons += layersize[i];
  conf->layersize = (int*)malloc((layersize[0]+1)*sizeof(int));
  for(i=0; i<layersize[0]+1; i++) conf->layersize[i] = layersize[i];

  // Compute the number of synapses:
  conf->nbsynapses = 0;
  for(i=1; i<layersize[0]; i++) conf->nbsynapses += layersize[i] * layersize[i+1]; 
  conf->nbsynapses *= 2;

  // Allocate the table of synapses:
  conf->synapses = (int*)malloc(2*conf->nbsynapses*sizeof(int));

  // creation of the synapses:
  int j,k=0,l,k2=0,k3=0;
  for(i=1;i<layersize[0];i++) {
    k3 += layersize[i];
    for(j=0; j<layersize[i]; j++) { 
      for(l=0; l<layersize[i+1]; l++) {
        // forward link/synapse:
        conf->synapses[k] = k2+j;
        k++;
        conf->synapses[k] = k3+l;
        k++;
        // Recurrent link/synapse:
        conf->synapses[k] = k3+l;
        k++;
        conf->synapses[k] = k2+j;
        k++;

      }
    }
    k2 += layersize[i];
  }
  return conf;
}

void freeconfig(CONFIG* conf) {
  free(conf->synapses);
  free(conf->layersize);
  free(conf);
}



RNN * creaternn(CONFIG * conf) {

  RNN * net = (RNN*)malloc(sizeof(RNN));
  net->nbneurons = conf->nbneurons;
  net->layersize = (int*)malloc((conf->layersize[0]+1)*sizeof(int));
  int i;
  for(i=0; i<conf->layersize[0]+1; i++) net->layersize[i] = conf->layersize[i];

  // Allocate the neuron table of the Recurrent Neural Network:
  net->n = (NEURON*)malloc(conf->nbneurons*sizeof(NEURON));

  // Initialize some neuron values:
  int j=0,k=0;
  for(i=0; i<conf->nbneurons; i++) {
    if(k==0) { k = conf->layersize[j+1]; j++; }
    net->n[i].layer = j-1;
    net->n[i].nbsynapsesin = 0; 
    k--;
  }

  // Count the incoming synapses for each neuron:
  k=0;
  for(i=0; i<conf->nbsynapses; i++) {
    k++;
    net->n[conf->synapses[k]].nbsynapsesin++;
    k++;
  }

  // Allocate weight table in neurons, and the table of pointer to neuron
  // that represent the incoming synapses:
  for(i=0; i<conf->nbneurons; i++) {
    net->n[i].weight = (double*)malloc(net->n[i].nbsynapsesin*sizeof(double));
    net->n[i].synapsesin = (NEURON**)malloc(net->n[i].nbsynapsesin*sizeof(NEURON*));
    net->n[i].nbsynapsesin = 0;
  }

  // Link the incoming synapses with the neurons:
  k=0;
  for(i=0; i<conf->nbsynapses; i++) {
    k++;
    net->n[conf->synapses[k]].synapsesin[net->n[conf->synapses[k]].nbsynapsesin] = &(net->n[conf->synapses[k-1]]);
    net->n[conf->synapses[k]].nbsynapsesin++;
    k++;
  }

  // Initialization of the values, errors, and weights:
  for(i=0; i<net->nbneurons; i++) {
    for(j=0; j<net->n[i].nbsynapsesin; j++) {
      net->n[i].weight[j] = 1.0 * (double)rand() / RAND_MAX - 1.0/2;
    }
    net->n[i].bias = 1.0 * (double)rand() / RAND_MAX - 1.0/2;
    net->n[i].value = 0.0;
    net->n[i].value_prev = 0.0;
    net->n[i].error_prev = 0.0;
    net->n[i].error = 0.0;
  }

  return net;
}


void freernn(RNN * net) {
  int i;
  for(i=0; i<net->nbneurons; i++) {
    free(net->n[i].weight);
    free(net->n[i].synapsesin);
  }
  free(net->n);
  free(net->layersize);
  free(net);
}

void rnnget(RNN * net, double * out) {
  int i,k=0;
  // Store the output of the network in the variable table "out":
  for(i=net->nbneurons-1; i>=(net->nbneurons - net->layersize[net->layersize[0]]); i--) { out[k] = net->n[i].value; k++; }
}

void rnnsetstart(RNN * net, double *input_matrix, double *output_matrix, double *d_weight, int m, int n, int k, size_t size_in_bytes) {
    // Allocate device memory for input and output matrices
    double *d_input, *d_output;
    hipMalloc(&d_input, size_in_bytes);  // Allocate memory for input matrix
    hipMalloc(&d_output, size_in_bytes); // Allocate memory for output matrix

    // Copy input matrix to device memory
    hipMemcpy(d_input, input_matrix, size_in_bytes, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((k + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    matrixMultiplicationKernel<<<gridSize, blockSize>>>(d_input, d_weight, d_output, m, n, k);

    // Copy the result back to host memory
    hipMemcpy(output_matrix, d_output, size_in_bytes, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

void rnnset(RNN * net, double * in) {
  int i,j,k;
  double v;

  NEURON *ni,*nj;
  // For each neuron:
  for(i=0; i<net->nbneurons; i++) {
    ni = &(net->n[i]);
    // If it is an input neuron:
    if(i<net->layersize[1]) ni->value = in[i];
    else ni->value = ni->bias;

    // If the neuron is NOT in input layer, then  
    // compute the value from the incoming synapses:
    if(i>=net->layersize[1]) {
      // For each incoming synapse:
      for(j=0; j<ni->nbsynapsesin; j++) {
        nj = ni->synapsesin[j];
        // If the synapse is from input layer to output layer, then tanh the value:
        if(nj->layer == 0 && ni->layer == (net->layersize[0]-1)) {
          ////////////////////////////////////////////////////////////////////////
          // Uncomment the following line to enable reccurent links computation:
          ni->value += tanh(nj->value_prev) * ni->weight[j];
          ////////////////////////////////////////////////////////////////////////
        } else {
          // If it is a forward link/synapse:
          if(ni->layer > nj->layer) ni->value +=  nj->value * ni->weight[j];
          ////////////////////////////////////////////////////////////////////////
          // Uncomment the following line to enable reccurent links computation:
          else ni->value += nj->value_prev * ni->weight[j];
          ////////////////////////////////////////////////////////////////////////
        }
      }
    }
    // If NOT the input layer NOR the output layer, then tanh the value:
    if(ni->layer != 0 && ni->layer != net->layersize[0]-1) ni->value = tanh(ni->value);
  }
}


void rnnlearnstart(RNN * net) {
  int i;
  // For each neuron, initialize error_prev and value_prev for a
  // new training cycle:
  for(i=0; i<net->nbneurons; i++) { net->n[i].error_prev = 0.0; net->n[i].value_prev = 0.0; }
}

void rnnlearn(RNN * net, double * out, double learningrate) {
  int i,j,k;
  k=0;

  NEURON *ni,*nj;
  // Initialize error to zero for the output layer:
  for(i=net->nbneurons-1; i>=net->nbneurons-net->layersize[net->layersize[0]]; i--) net->n[i].error = 0.0;

  // Compute the error for output neurons, and 
  // initialize it to 0 for the other neurons:
  for(i=net->nbneurons-1; i>=0; i--) {
    ni = &(net->n[i]);
    // If ni is an output neuron, update the error:
    if(ni->layer == net->layersize[0]-1) {
      ni->error += ni->value - out[k];
      k++;
    } else {
      ni->error = 0.0;
    }
  }

  // Compute error for all other neurons:
  for(i=net->nbneurons-1; i>=0; i--) {
    ni = &(net->n[i]);
    // For each incoming synapse NOT from output layer:
    for(j=0; j<ni->nbsynapsesin; j++) {
      nj = ni->synapsesin[j];
      // If it is a forward link/synapse:
      if(ni->layer > nj->layer) nj->error += ni->error * ni->weight[j];
    }
  }

  // Update weights:
  for(i=0; i<net->nbneurons; i++) {
    ni = &(net->n[i]);
    double wchange,derivative;
    // For the output layer:
    if(ni->layer == net->layersize[0]-1) {
      derivative = ni->error * learningrate;
      // For each incoming synapse:
      for(j=0; j<ni->nbsynapsesin; j++) {
        nj = ni->synapsesin[j];
        wchange = derivative;
        // If it is a forward link/synapse:
        if(ni->layer > nj->layer) wchange *= nj->value;
        else wchange *= nj->value_prev;
        ni->weight[j] -= wchange;
        if(ni->weight[j] > 5) ni->weight[j] = 5;
        if(ni->weight[j] < -5) ni->weight[j] = -5;
      }
      ni->bias -= derivative;
      if(ni->bias > 5) ni->bias = 5;
      if(ni->bias < -5) ni->bias = -5;

    // For the other layers:
    } else {
      derivative = 1.0 - ni->value * ni->value;
      derivative *= ni->error * learningrate;
      // For each incoming synapse:
      for(j=0; j<ni->nbsynapsesin; j++) {
        nj = ni->synapsesin[j];
        wchange = derivative;
        // If it is a forward link/synapse:
        if(ni->layer > nj->layer) wchange *= nj->value;
        else wchange *= nj->value_prev;
        ni->weight[j] -= wchange;
      }
      ni->bias -= derivative;
    }
  }

  // Update error_prev:
  for(i=0; i<net->nbneurons; i++) net->n[i].error_prev = net->n[i].error;
}

int main() {
    srand(time(NULL));

    // Déclarations des variables pour les matrices et les poids
    float *input_matrix, *output_matrix, *d_weight;
    int k, m, n;
    size_t taille_input_matrix = 16;
    size_t taille_output_matrix = 16;
    size_t taille_d_weight = 16;

    // Allocation de mémoire pour les matrices et les poids sur le GPU
    hipMalloc((void**)&input_matrix, taille_input_matrix);
    hipMalloc((void**)&output_matrix, taille_output_matrix);
    hipMalloc((void**)&d_weight, taille_d_weight);

    int layersize_netrnn[] = { 4, 1, 25, 12, 1 };
    CONFIG * configrnn = createconfig(layersize_netrnn);
    RNN * netrnn = creaternn(configrnn);

    double inc,outc;
    double global_error2 = 1;
    int i2=0;
    int iter;

    //////////////////////////////////////////////////////
    // Training of the Recurrent Neural Network:
    //////////////////////////////////////////////////////

    // Votre boucle d'entraînement ici...

    // Libération de la mémoire allouée sur le GPU
    hipFree(input_matrix);
    hipFree(output_matrix);
    hipFree(d_weight);

    // Libération de la mémoire allouée dynamiquement
    freeconfig(configrnn);
    freernn(netrnn);
    
    return 0;
}
