#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

float learning_rate = 0.0001;
int seq_len = 50;
int max_epochs = 25;
int hidden_dim = 100;
int output_dim = 1;
int bptt_truncate = 5;
int min_clip_val = -10;
int max_clip_val = 10;



double sigmoid(float x){
	return 1/(1+exp(-x))
};


double mean_squared_error(double *y_true, double *y_pred, int len) {
    double mse = 0.0;
    for (int i = 0; i < len; i++) {
        double error = y_true[i] - y_pred[i];
        mse += error * error;       
    }
    return mse / len;
}


double calculate_loss(double **X, double **Y, double **U, double **V, double **W, double *loss_, double *activation_) {

    double loss = 0.0;

    for (int i = 0; i < num_records; i++) { //

        double *x = X[i];
        double *y = Y[i];

        double *prev_activation = (double *)malloc(hidden_dim * sizeof(double));

        for (int j = 0; j < hidden_dim; j++) {
            prev_activation[j] = 0.0;
        }

        for (int timestep = 0; timestep < seq_len; timestep++) {

            double *new_input = (double *)malloc(seq_len * sizeof(double));

            for (int k = 0; k < seq_len; k++) {
                new_input[k] = 0.0;
            }

            new_input[timestep] = x[timestep];


            double *mulu = (double *)malloc(hidden_dim * sizeof(double));
            double *mulw = (double *)malloc(hidden_dim * sizeof(double));


            for (int k = 0; k < hidden_dim; k++) {
                mulu[k] = 0.0;
                for (int l = 0; l < seq_len; l++) {
                    mulu[k] += U[k][l] * new_input[l];
                }
            }

            for (int k = 0; k < hidden_dim; k++) {
                mulw[k] = 0.0;
                for (int l = 0; l < hidden_dim; l++) {
                    mulw[k] += W[k][l] * prev_activation[l];
                }
            }

            double _sum = 0.0;

            for (int k = 0; k < hidden_dim; k++) {
                _sum += mulu[k] + mulw[k];
            }

            
            double activation[hidden_dim];

            for (int k = 0; k < hidden_dim; k++) {
                activation[k] = sigmoid(_sum);
            }

            double *mulv = (double *)malloc(output_dim * sizeof(double));

            for (int k = 0; k < output_dim; k++) {
                mulv[k] = 0.0;
                for (int l = 0; l < hidden_dim; l++) {
                    mulv[k] += V[k][l] * activation[l];
                }
            }

            for (int k = 0; k < hidden_dim; k++) {
                prev_activation[k] = activation[k];
            }

        double loss_per_record = (y - mulv[0]) * (y - mulv[0]) / 2.0;
        loss += loss_per_record;

        }
    }
 
    *loss_ = loss;
    *activation_ = activation;


    return 0;
}



typedef struct {
    double *activation;
    double *prev_activation;
} Layer;


Layer *calc_layers(double **x, double **U, double **V, double **W, double *prev_activation) {
    
	Layer *layers = (Layer *)malloc(seq_len * sizeof(Layer));
    
    double *mulu = (double *)malloc(hidden_dim * sizeof(double));
    double *mulv = (double *)malloc(output_dim * sizeof(double));
    double *mulw = (double *)malloc(hidden_dim * sizeof(double));

    for (int timestep = 0; timestep < seq_len; timestep++) {

        double *new_input = (double *)malloc(seq_len * sizeof(double));
        for (int k = 0; k < seq_len; k++) {
            new_input[k] = 0.0;
        }

        new_input[timestep] = x[timestep];


        for (int k = 0; k < hidden_dim; k++) {
            mulu[k] = 0.0;
            for (int l = 0; l < seq_len; l++) {
                mulu[k] += U[k][l] * new_input[l];
            }
        }

        for (int k = 0; k < hidden_dim; k++) {
            mulw[k] = 0.0;
            for (int l = 0; l < hidden_dim; l++) {
                mulw[k] += W[k][l] * prev_activation[l];
            }
        }

        double _sum = 0.0;

        for (int k = 0; k < hidden_dim; k++) {
            _sum += mulu[k] + mulw[k];
        }

            
        double activation[hidden_dim];
        for (int k = 0; k < hidden_dim; k++) {
            activation[k] = sigmoid(_sum);
        }

        double *mulv = (double *)malloc(output_dim * sizeof(double));

        for (int k = 0; k < output_dim; k++) {
            mulv[k] = 0.0;
            for (int l = 0; l < hidden_dim; l++) {
                mulv[k] += V[k][l] * activation[l];
            }
        }


        layers[timestep].activation = (double *)malloc(hidden_dim * sizeof(double));
        layers[timestep].prev_activation = (double *)malloc(hidden_dim * sizeof(double));
        
        for (int i = 0; i < hidden_dim; i++) {
            layers[timestep].activation[i] = activation[i];
            layers[timestep].prev_activation[i] = prev_activation[i];
        }

        // Update prev_activation for the next timestep
        for (int k = 0; k < hidden_dim; k++) {
            prev_activation[k] = activation[k];
        }

    }


    return layers;
}







double **backprop(double **x, double **U, double **V, double **W, double *dmulv, double **mulu, double **mulw, Layer *layers) {



    double **dU = (double **)malloc(hidden_dim * sizeof(double *));
    double **dV = (double **)malloc(output_dim * sizeof(double *));
    double **dW = (double **)malloc(hidden_dim * sizeof(double *));
    
    double **dU_t = (double **)malloc(hidden_dim * sizeof(double *));
    double **dW_t = (double **)malloc(hidden_dim * sizeof(double *)); 
    


    for (int i = 0; i < hidden_dim; i++) {
        dU[i] = (double *)malloc(seq_len * sizeof(double));
        dW[i] = (double *)malloc(hidden_dim * sizeof(double));
        
        dU_t[i] = (double *)malloc(seq_len * sizeof(double));
        dW_t[i] = (double *)malloc(hidden_dim * sizeof(double));
        
    }
    
    for (int i = 0; i < output_dim; i++) {
        dV[i] = (double *)malloc(hidden_dim * sizeof(double));
    }

    for (int i = 0; i < hidden_dim; i++) {
        for (int j = 0; j < seq_len; j++) {
            dU[i][j] = 0.0;
            dU_t[i][j] = 0.0;
            dU_i[i][j] = 0.0;
        }
        for (int j = 0; j < hidden_dim; j++) {
            dW[i][j] = 0.0;
            dW_t[i][j] = 0.0;
        }
    }

    for (int i = 0; i < output_dim; i++) {
        for (int j = 0; j < hidden_dim; j++) {
            dV[i][j] = 0.0;;
        }
    }



    // Calculation


    double _sum;
    _sum = **mulu + **mulw;

    double **dsv = (double **)malloc(hidden_dim * sizeof(double *)); 

    for (int i = 0; i < hidden_dim; i++) {
        dsv[i] = (double *)malloc(hidden_dim * sizeof(double));
        for (int j = 0; j < hidden_dim; j++) {
            dsv[i][j] = 0;
            for (int k = 0; k < hidden_dim; k++) {
                dsv[i][j] += V[k][i] * dmulv[k];
            }
        }
    }


    double *get_previous_activation_differential(double _sum, double *ds, double **W) {
        
        double *d_sum = (double *)malloc(hidden_dim * sizeof(double));
        
        for (int i = 0; i < hidden_dim; i++) {
            d_sum[i] = _sum * (1 - _sum) * ds[i];
        }


        double *dmulw = (double *)malloc(hidden_dim * sizeof(double));
        
        for (int i = 0; i < hidden_dim; i++) {
            dmulw[i] = d_sum[i] * 1.0; // Ici, l'opération `np.ones_like(ds)` en Python est remplacée par 1.0 en C
        }


        double **result = (double **)malloc(hidden_dim * sizeof(double *)); 
        // Produit matriciel entre la transposée de W et dmulw
        for (int i = 0; i < hidden_dim; i++) {
            result[i] = (double *)malloc(hidden_dim * sizeof(double));
            for (int j = 0; j < hidden_dim; j++) {
                result[i][j] = 0;
                for (int k = 0; k < hidden_dim; k++) {
                    result[i][j] += W[k][i] * dmulw[k];
                }
            }
        }

        return result;
    }



    for (int timestep = 0; timestep < seq_len; timestep++) {
        
        double **dV_t = (double **)malloc(output_dim * sizeof(double *)); 
        for (int i = 0; i < output_dim; i++) {
            dV_t[i] = (double *)malloc(hidden_dim * sizeof(double));
            for (int j = 0; j < hidden_dim; j++) {
                dV_t[i][j] = 0;
                for (int k = 0; k < hidden_dim; k++) {
                    dV_t[i][j] += layers[timestep]['prev_activation'][k][i] * dmulw[k];
                }
            }
        }

        double ds = dsv;

        double **dprev_activation = get_previous_activation_differential(_sum, dsv, W);

        
        
        for (int k = timestep - 1; k >= fmax(-1, timestep - bptt_truncate - 1); k--) {
            
            for (int i = 0; i < hidden_dim; i++) {
                dsv[i] += dprev_activation[i];
            }

            double **dprev_activation = get_previous_activation_differential(_sum, dsv, W);
            
            double **dW_i = (double **)malloc(hidden_dim * sizeof(double *)); 

            for (int i = 0; i < hidden_dim; i++) {
                dW_i[i] = (double *)malloc(hidden_dim * sizeof(double));
                for (int j = 0; j < hidden_dim; j++) {
                    dW_i[i][j] = 0;
                    for (int k = 0; k < hidden_dim; k++) {
                        dW_i[i][j] += W[i][k] * layers[timestep]['prev_activation'][k][i];
                    }
                }
            }

            double *new_input = (double *)malloc(seq_len * sizeof(double));
            for (int i = 0; i < seq_len; i++) {
                new_input[i] = 0.0;
            }
            new_input[timestep] = x[timestep];


            double **dU_i = (double **)malloc(hidden_dim * sizeof(double *)); 
                
            for (int i = 0; i < hidden_dim; i++) {
                dU_i[i] = (double *)malloc(seq_len * sizeof(double));
                for (int j = 0; j < seq_len; j++) {
                    dU_i[i][j] = 0;
                    for (int k = 0; k < seq_len; k++) {
                        dU_i[i][j] += U[i][k] * new_input[k];
                    }
                }
            }


            for (int i = 0; i < hidden_dim; i++) {
                for (int j = 0; j < seq_len; j++) {
                    dU_t[i][j] += dU_i[i][j];
                    dW_t[i][j] += dW_i[i][j];
                }
            }
        }

        for (int i = 0; i < hidden_dim; i++) {
            for (int j = 0; j < seq_len; j++) {
                dU[i][j] += dU_t[i][j];
                dW[i][j] += dW_t[i][j];
            }
        }

    }


    // exploding gradients
    for (int i = 0; i < hidden_dim; i++) {
        for (int j = 0; j < seq_len; j++) {
            if (dU[i][j] > max_clip_val) {
                dU[i][j] = max_clip_val;
            }
            if (dU[i][j] < min_clip_val) {
                dU[i][j] = min_clip_val;
            }
        }
        for (int j = 0; j < hidden_dim; j++) {
            if (dW[i][j] > max_clip_val) {
                dW[i][j] = max_clip_val;
            }
            if (dW[i][j] < min_clip_val) {
                dW[i][j] = min_clip_val;
            }
        }
    }

    for (int i = 0; i < output_dim; i++) {
        for (int j = 0; j < hidden_dim; j++){
            if (dV[i][j] > max_clip_val) {
                dV[i][j] = max_clip_val;
            }
            if (dV[i][j] < min_clip_val) {
                dV[i][j] = min_clip_val;
            }
        }
    }

    return 0;
}





double **train(double **U, double **V, double **W, double **X, double **Y, double **X_validation, double **Y_validation) {
    

    for (int epoch = 0; epoch < max_epochs; epoch++) {
       
        double loss_training,preactivation_training;
        calculate_loss(X, Y, U, V, W, &loss_training, &preactivation_training);
        
        double loss_validation , _ ;
        calculate_loss(X_validation, Y_validation, U, V, W, &loss_validation, &_);


        printf("Epoch: %d, Loss: %f, Validation Loss: %f\n", epoch+1, loss, val_loss);


        for (int i = 0; i < Y.shape[0]; i++) {
            double **x = X[i];
            double **y = Y[i];

            double *prev_activation = (double *)malloc(hidden_dim * sizeof(double));
            for (int j = 0; j < hidden_dim; j++) {
                prev_activation[j] = 0.0;
            }
            
            layers = calc_layers(x, U, V, W, prev_activation);


            double **dmulv = (double **)malloc(hidden_dim * sizeof(double *));

            for (int j = 0; j < hidden_dim; j++) {
                dmulv[j] = (double *)malloc(sizeof(double));
                dmulv[j][0] = mulv[j][0] - y[j][0];
            }


            backprop(x, U, V, W, dmulv, mulu, mulw, layers);


            for (int j = 0; j < hidden_dim; j++) {
                for (int k = 0; k < seq_len; k++) {
                    U[j][k] -= learning_rate * dU[j][k];
                    V[j][k] -= learning_rate * dV[j][k];
                    W[j][k] -= learning_rate * dW[j][k];
                }
            }


        }
    }

    return 0; 
}








int main () {

    double sin_wave[200];

    for (int i = 0; i < 200; i++) {
        sin_wave[i] = sin(i);
    }

    int num_records = 200 - seq_len;
    double **X, **Y;

    X = (double **)malloc(num_records * sizeof(double *));
    Y = (double **)malloc(num_records * sizeof(double *));

    for (int i = 0; i < num_records - 50; i++) {

        X[i] = (double *)malloc(seq_len * sizeof(double));
        Y[i] = (double *)malloc(sizeof(double));

        for (int j = 0; j < seq_len; j++) {
            X[i][j] = sin_wave[i + j];
        }
        Y[i][0] = sin_wave[i + seq_len];
    }

    double  **X_validation, **Y_validation;

    X_validation = (double **)malloc(50 * sizeof(double *));
    Y_validation = (double **)malloc(50 * sizeof(double *));

    for (int i = num_records - seq_len; i < num_records; i++) {
        X_validation[i - num_records + seq_len] = (double *)malloc(seq_len * sizeof(double));
        Y_validation[i - num_records + seq_len] = (double *)malloc(sizeof(double));

        for (int j = 0; j < seq_len; j++) {
            X_validation[i - num_records + seq_len][j] = sin_wave[i + j];
        }
        Y_validation[i - num_records + seq_len][0] = sin_wave[i + seq_len];
    }


    srand(12161);
  
    double **U;
    U = (double **)malloc(hidden_dim * sizeof(double *));
    for (int i = 0; i < hidden_dim; i++) {
        U[i] = (double *)malloc(seq_len * sizeof(double));
        for (int j = 0; j < seq_len; j++) {
            U[i][j] = (double)rand() / RAND_MAX;
        }
    }
    
    double **V;
    V = (double **)malloc(output_dim * sizeof(double *));
    for (int i = 0; i < output_dim; i++) {
        V[i] = (double *)malloc(hidden_dim * sizeof(double));
        for (int j = 0; j < hidden_dim; j++) {
            V[i][j] = (double)rand() / RAND_MAX;
        }
    }

    double **W;
    W = (double **)malloc(hidden_dim * sizeof(double *));
    for (int i = 0; i < hidden_dim; i++) {
        W[i] = (double *)malloc(hidden_dim * sizeof(double));
        for (int j = 0; j < hidden_dim; j++) {
            W[i][j] = (double)rand() / RAND_MAX;
        }
    }

    // Train the RNN
    
    train(U, V, W, X, Y, X_validation, Y_validation);




   //predictions on the training set

   double **predictions = (double **)malloc(num_records * sizeof(double *));
   for (int i = 0; i < num_records; i++) {
        predictions[i] = (double *)malloc(output_dim * sizeof(double));
   }

    for (int i = 0; i < num_records; i++) {
        double *x = X[i];
        double y = Y[i][0];
        double *prev_activation = (double *)malloc(hidden_dim * sizeof(double));
        for (int i = 0; i < hidden_dim; i++) {
            prev_activation[i] = 0.0;
        }
        //memset(prev_activation, 0, hidden_dim * sizeof(double)); // Initialisation à zéro

        for (int timestep = 0; timestep < seq_len; timestep++) {

            double mulu = 0.0;
            for (int j = 0; j < seq_len; j++) {
                mulu += U[j][i] * x[j];
            }
        
            double mulw = 0.0;
            for (int j = 0; j < hidden_dim; j++) {
                mulw += W[j][i] * prev_activation[j];
            }
        
            double _sum = mulu + mulw;
        
            double activation = sigmoid(_sum);
        
            double mulv = 0.0;
            for (int j = 0; j < hidden_dim; j++) {
                mulv += V[j][i] * activation;
            }
        
            for (int j = 0; j < hidden_dim; j++) {
            prev_activation[j] = activation;
            }
        }

        for (int j = 0; j < output_dim; j++) {
            predictions[i][j] = mulv;
        }
    }


   //predictions on the training set

   double **predictions = (double **)malloc(num_records * sizeof(double *));
   for (int i = 0; i < num_records; i++) {
        predictions[i] = (double *)malloc(output_dim * sizeof(double));
   }

    for (int i = 0; i < num_records; i++) {
        double *x = X[i];
        double y = Y[i][0];
        double *prev_activation = (double *)malloc(hidden_dim * sizeof(double));
        for (int i = 0; i < hidden_dim; i++) {
            prev_activation[i] = 0.0;
        }
        //memset(prev_activation, 0, hidden_dim * sizeof(double)); // Initialisation à zéro

        for (int timestep = 0; timestep < seq_len; timestep++) {

            double mulu = 0.0;
            for (int j = 0; j < seq_len; j++) {
                mulu += U[j][i] * x[j];
            }
        
            double mulw = 0.0;
            for (int j = 0; j < hidden_dim; j++) {
                mulw += W[j][i] * prev_activation[j];
            }
        
            double _sum = mulu + mulw;
        
            double activation = sigmoid(_sum);
        
            double mulv = 0.0;
            for (int j = 0; j < hidden_dim; j++) {
                mulv += V[j][i] * activation;
            }
        
            for (int j = 0; j < hidden_dim; j++) {
            prev_activation[j] = activation;
            }
        }

        for (int j = 0; j < output_dim; j++) {
            predictions[i][j] = mulv;
        }
   }


   //predictions on the validation set

   double **val_predictions = (double **)malloc(num_records * sizeof(double *));
   for (int i = 0; i < num_records; i++) {
        val_predictions[i] = (double *)malloc(output_dim * sizeof(double));
   }

    for (int i = 0; i < num_records; i++) {
        double *x = X[i];
        double y = Y[i][0];
        double *prev_activation = (double *)malloc(hidden_dim * sizeof(double));
        for (int i = 0; i < hidden_dim; i++) {
            prev_activation[i] = 0.0;
        }
        //memset(prev_activation, 0, hidden_dim * sizeof(double)); // Initialisation à zéro

        for (int timestep = 0; timestep < seq_len; timestep++) {

            double mulu = 0.0;
            for (int j = 0; j < seq_len; j++) {
                mulu += U[j][i] * x[j];
            }
        
            double mulw = 0.0;
            for (int j = 0; j < hidden_dim; j++) {
                mulw += W[j][i] * prev_activation[j];
            }
        
            double _sum = mulu + mulw;
        
            double activation = sigmoid(_sum);
        
            double mulv = 0.0;
            for (int j = 0; j < hidden_dim; j++) {
                mulv += V[j][i] * activation;
            }
        
            for (int j = 0; j < hidden_dim; j++) {
            prev_activation[j] = activation;
            }
        }

        for (int j = 0; j < output_dim; j++) {
            val_predictions[i][j] = mulv;
        }
    }


    //double mse = mean_squared_error(Y_validation, val_predictions, size);
    //double rmse = sqrt(mse);
    // Affichage du RMSE
    //printf("%f\n", rmse);
    
    return 0;

};
