#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

typedef float basetype;  // Utilisation du type float pour les données

typedef struct resnfo {
    int seconds;        // Secondes
    int microseconds;   // Microsecondes
    // D'autres membres peuvent être ajoutés au besoin
} resnfo;

void timestamp(struct timespec *ts) {
    clock_gettime(CLOCK_MONOTONIC, ts);
}

// Fonction de multiplication de matrices sur le CPU
void MultMat_CPU(const basetype arrayA[], const basetype arrayB[], 
                 basetype arrayR[], const unsigned int n)
{
    unsigned int i, j, k;
    basetype res;

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            res = 0;
            for (k = 0; k < n; k++) {
                res += arrayA[i * n + k] * arrayB[k * n + j];
            }
            arrayR[i * n + j] = res;
        }
    }
}

// Fonction de multiplication de matrices sur le GPU
__global__ void multmat_kernel_cuda(const basetype *const mA, 
                                    const basetype *const mB, 
                                    basetype *const mR, const int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < n && col < n) {
        basetype Pvalue = 0;
        for (int k = 0; k < n; ++k) {
            Pvalue += mA[row * n + k] * mB[k * n + col];
        }
        mR[row * n + col] = Pvalue;
    }
}

void multmat_GPU(const basetype arrayA[], const basetype arrayB[], 
                 basetype arrayR[], const unsigned int n, 
                 const unsigned int blk_size, 
                 resnfo *const start, resnfo *const end)
{
    unsigned int numBytes = n * n * sizeof(basetype);

    basetype *cA, *cB, *cR;
    hipMalloc((void **)&cA, numBytes);
    hipMalloc((void **)&cB, numBytes);
    hipMalloc((void **)&cR, numBytes);

    hipMemcpy(cA, arrayA, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(cB, arrayB, numBytes, hipMemcpyHostToDevice);

    dim3 dimBlock(blk_size, blk_size);
    dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x, (n + dimBlock.y - 1) / dimBlock.y);

    timestamp(start);

    multmat_kernel_cuda<<<dimGrid, dimBlock>>>(cA, cB, cR, n);
    
    hipDeviceSynchronize();
    timestamp(end);

    hipMemcpy(arrayR, cR, numBytes, hipMemcpyDeviceToHost);

    hipFree(cA);
    hipFree(cB);
    hipFree(cR);
}

int main() {
    const int n = 4;  // Taille des matrices
    basetype matrixA[n * n], matrixB[n * n], result_CPU[n * n], result_GPU[n * n];

    // Initialisation des matrices A et B avec des valeurs aléatoires
    for (int i = 0; i < n * n; i++) {
        matrixA[i] = rand() % 10;  // Valeurs aléatoires entre 0 et 9
        matrixB[i] = rand() % 10;
    }

    // Appel de la fonction pour la multiplication de matrices sur le CPU
    MultMat_CPU(matrixA, matrixB, result_CPU, n);

    // Appel de la fonction pour la multiplication de matrices sur le GPU
    resnfo start, end;
    multmat_GPU(matrixA, matrixB, result_GPU, n, 16, &start, &end);

    // Affichage des résultats
    printf("Résultat de la multiplication de matrices sur le CPU :\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f ", result_CPU[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("Résultat de la multiplication de matrices sur le GPU :\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f ", result_GPU[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");

    return 0;
}
