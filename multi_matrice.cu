#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

typedef float basetype;  

// Fonctions de multiplication de matrices en C
void MultMat_CPU(const basetype arrayA[], const basetype arrayB[], 
      basetype arrayR[], const unsigned int n)
{
    unsigned int i, j, k;
    basetype res;

    for(i = 0; i < n; i++) 
        for(j= 0; j<n; j++){
            res = 0;
            for(k=0; k<n; k++)
                res += arrayA[i*n+k] * arrayB[k*n+j];
 
            arrayR[i*n+j]= res;
        }
}

__global__ void multmat_kernel_cuda(const basetype *const mA, 
      const basetype *const mB, 
      basetype *const mR, const int n)
{
    //2D Thread ID
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < n && col < n) {
        basetype Pvalue = 0;
        for (int k = 0; k < n; ++k) {
            Pvalue += mA[row*n + k] * mB[k*n + col];
        }
        mR[row*n + col] = Pvalue;
    }
}

void multmat_GPU(const basetype arrayA[], const basetype arrayB[], 
      basetype arrayR[], const unsigned int n, 
      const unsigned int blk_size)
{
    unsigned int numBytes = n * n * sizeof(basetype);

    basetype *cA, *cB, *cR;
    hipMalloc((void **)&cA, numBytes);
    hipMalloc((void **)&cB, numBytes);
    hipMalloc((void **)&cR, numBytes);

    hipMemcpy(cA, arrayA, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(cB, arrayB, numBytes, hipMemcpyHostToDevice);

    dim3 dimBlock(blk_size, blk_size);
    dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x, (n + dimBlock.y - 1) / dimBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    multmat_kernel_cuda<<<dimGrid, dimBlock>>>(cA, cB, cR, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Temps d'exécution sur le GPU : %f ms\n", milliseconds);

    hipMemcpy(arrayR, cR, numBytes, hipMemcpyDeviceToHost);

    hipFree(cA);
    hipFree(cB);
    hipFree(cR);
}

int main() {
    const int n = 4;  
    basetype matrixA[n * n], matrixB[n * n], result_CPU[n * n], result_GPU[n * n];

    // Initialisation des matrices A et B avec des valeurs aléatoires
    for (int i = 0; i < n * n; i++) {
        matrixA[i] = rand() % 10;  
        matrixB[i] = rand() % 10;
    }

    // Appel de la fonction pour la multiplication de matrices sur le CPU
    clock_t start_CPU = clock();
    MultMat_CPU(matrixA, matrixB, result_CPU, n);
    clock_t end_CPU = clock();
    printf("Temps d'exécution sur le CPU : %f ms\n", ((double)(end_CPU - start_CPU)) * 1000.0 / CLOCKS_PER_SEC);

    // Appel de la fonction pour la multiplication de matrices sur le GPU
    multmat_GPU(matrixA, matrixB, result_GPU, n, 16);

    // Affichage des résultats
    printf("Résultat de la multiplication de matrices sur le CPU :\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f ", result_CPU[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("Résultat de la multiplication de matrices sur le GPU :\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f ", result_GPU[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");

    return 0;
}
